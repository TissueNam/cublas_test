
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "hipblas.h"

#define M 5
#define N 3
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

void printMat(float* mat, int m, int n){
    for(int row = 0; row < m; row++){
        for(int col = 0; col < n; col++){
            printf("%.f ",mat[IDX2C(row, col, m)]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int row, col;
    float* h_A; // 5x3 vector
    float* h_X; // 3x1 matrix
    float* h_Y; // 5x1 vector
    float* d_A = 0;
    float* d_X = 0;
    float* d_Y = 0;

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Allocating host memory
    h_A = (float*)malloc(M*N*sizeof(*h_A));
    h_X = (float*)malloc(N*1*sizeof(*h_X));
    h_Y = (float*)malloc(M*1*sizeof(*h_Y));
    if (!h_A || !h_X || !h_Y){
        printf("host memory allocation failed\n");
        return EXIT_FAILURE;
    }

    // Initilize host memory 
    for (col = 0; col < N; col++){
        for(row = 0; row < M; row++){
            h_A[IDX2C(row, col, M)] = (float)(row*N + col + 1);
        }
    }
    for (row = 0 ; row < N ; row++){
        h_X[row] = (float)(row+1);
    }
    // printMat(h_X, N, 1);
    // for (int i = 0; i < M*N ; i++){
    //     printf("%.f ", h_A[i]);
    // }
    // printf("\n");
    // printf("host A(%dx%d) = \n", M, N);
    // printMat(h_A, M, N);

    // Allocating device memory
    cudaStat = hipMalloc((void**)&d_A, M*N*sizeof(float));
    cudaStat = hipMalloc((void**)&d_X, N*1*sizeof(float));
    cudaStat = hipMalloc((void**)&d_Y, M*1*sizeof(float));
    if (cudaStat != hipSuccess){
        printf("device memory allocation failed\n");
        return EXIT_FAILURE;
    }

    // Initialize CUBLAS & Create Handle
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS){
        printf("CUBLAS initilization failed\n");
        return EXIT_FAILURE;
    }

    // Copying matrix values to device memory
    stat = hipblasSetMatrix(M, N, sizeof(float), h_A, M, d_A, M);
    // stat = cublasSetMatrix(N, 1, sizeof(float), h_X, N, d_X, N);
    // stat = cublasSetMatrix(M, 1, sizeof(float), h_Y, M, d_Y, M);
    stat = hipblasSetVector(N, sizeof(float), h_X, 1, d_X, 1);
    stat = hipblasSetVector(M, sizeof(float), h_Y, 1, d_Y, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed\n");
        hipFree(d_A);
        hipFree(d_X);
        hipFree(d_Y);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    // Run cublas 32bit integer type gemm
    stat = hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, d_X, 1, &beta, d_Y, 1);
    printf("stat: %d\n", stat);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("cublas gemv error\n");
        hipFree(d_A);
        hipFree(d_X);
        hipFree(d_Y);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    // copy result from device to host
    stat = hipblasGetVector(M, sizeof(float), d_Y, 1, h_Y, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree(d_A);
        hipFree(d_X);
        hipFree(d_Y);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipFree(d_A);
    hipFree(d_X);
    hipFree(d_Y);

    hipblasDestroy(handle);

    // Print ouput
    printf("host Y(%dx%d) = \n", M, 1);
    printMat(h_Y, M, 1);

    free(h_A);
    free(h_X);
    free(h_Y);

    return EXIT_SUCCESS;
}
